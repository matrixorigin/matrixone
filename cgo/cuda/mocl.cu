
#include <hip/hip_runtime.h>
#include <stdint.h>

// Device code
extern "C" __global__ void l2distance_f32(
        double *res, int n, int vecsz, bool sq,
        const uint32_t *offlenA, const uint8_t *A, 
        const uint32_t *offlenB, const uint8_t *B) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int loop = vecsz / sizeof(float);
    if (i < n) {
        res[i] = 0;
        uint32_t offA = offlenA[i * 6 + 1];
        uint32_t offB = offlenB[i * 6 + 1];
        float *astart = (float *)(A + offA);
        float *bstart = (float *)(B + offB); 
        for (int j = 0; j < loop; j++) {
            float diff = astart[j] - bstart[j];
            res[i] += diff * diff;
        }
        if (!sq) {
            res[i] = sqrt(res[i]);
        }
    }
}

extern "C" __global__ void l2distance_f32_const(
        double *res, int n, int vecsz, bool sq,
        const uint32_t *offlenA, const uint8_t *A,
        const float *B) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int loop = vecsz / sizeof(float);
    if (i < n) {
        res[i] = 0;
        uint32_t offA = offlenA[i * 6 + 1];
        float *astart = (float *)(A + offA);
        for (int j = 0; j < loop; j++) {
            float diff = astart[j] - B[j];
            res[i] += diff * diff;
        }
        if (!sq) {
            res[i] = sqrt(res[i]);
        }
    }
}

extern "C" __global__ void l2distance_f64(
        double *res, int n, int vecsz, bool sq,
        const uint32_t *offlenA, const uint8_t *A, 
        const uint32_t *offlenB, const uint8_t *B) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int loop = vecsz / sizeof(double);
    if (i < n) {
        res[i] = 0;
        uint32_t offA = offlenA[i * 6 + 1];
        uint32_t offB = offlenB[i * 6 + 1];
        double *astart = (double *)(A + offA);
        double *bstart = (double *)(B + offB); 
        for (int j = 0; j < loop; j++) {
            float diff = astart[j] - bstart[j];
            res[i] += diff * diff;
        }
        if (!sq) {
            res[i] = sqrt(res[i]);
        }
    }
}

extern "C" __global__ void l2distance_f64_const(
        double *res, int n, int vecsz, bool sq,
        const uint32_t *offlenA, const uint8_t *A,
        const double *B) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int loop = vecsz / sizeof(double);
    if (i < n) {
        res[i] = 0;
        uint32_t offA = offlenA[i * 6 + 1];
        double *astart = (double *)(A + offA);
        for (int j = 0; j < loop; j++) {
            float diff = astart[j] - B[j];
            res[i] += diff * diff;
        }
        if (!sq) {
            res[i] = sqrt(res[i]);
        }
    }
}
